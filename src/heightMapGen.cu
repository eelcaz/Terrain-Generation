#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include "terrain_generator.h"

__device__ double interpolate(double a, double b, double weight) {
    if (weight < 0) return a;
    if (weight > 1) return b;
    return (b - a) * ((weight * (weight * 6.0 - 15.0) + 10.0) * weight * weight * weight) + a;
};

__device__ double dotProduct(int GridZ, int GridX, double pz, double px, int* permutation) {
    // get the random vector on the gridPoint
    int randDir = permutation[(permutation[abs(GridZ) % 256] + abs(GridX)) % 256];
    double gradZ = cos((double)randDir);
    double gradX = sin((double)randDir);
    // get the offset vector from the grid point to the target point
    double offsetZ = pz-(double)GridZ;
    double offsetX = px-(double)GridX;

    return gradZ * offsetZ + gradX * offsetX;
};

__global__ void chunkHeightMapKernel(int chunkZ, int chunkX, int* heightMap, int* permutation) {
    double offset = (double)1/(2*(Terrain::CHUNK_WIDTH-1));
    int _z = threadIdx.x / Terrain::CHUNK_WIDTH;
    int _x = threadIdx.x % Terrain::CHUNK_WIDTH;
    double z = (chunkZ + offset + (double)_z/(Terrain::CHUNK_WIDTH-1))/Terrain::TERRAIN_ZOOM;
    double x = (chunkX + offset + (double)_x/(Terrain::CHUNK_WIDTH-1))/Terrain::TERRAIN_ZOOM;
    double noiseZ, noiseX;

    // fbm iterations
    int octaves = 6;
    double total = 0.0;
    double maxVal = 0;
    for (int i = 0; i < octaves; ++i) {
        double amplitude = pow(0.58, (double) i);
        double frequency = pow(2.0, (double) i);

        noiseZ = z * frequency;
        noiseX = x * frequency;

        // noise calculations
        int zGrid0 = (int)floor(noiseZ);
        int xGrid0 = (int)floor(noiseX);
        int zGrid1 = zGrid0 + 1;
        int xGrid1 = xGrid0 + 1;

        // calculate weights
        double wz = noiseZ - (double)zGrid0;
        double wx = noiseX - (double)xGrid0;

        double dot1, dot2, interp1, interp2;
        dot1 = dotProduct(zGrid0, xGrid0, noiseZ, noiseX, permutation);
        dot2 = dotProduct(zGrid1, xGrid0, noiseZ, noiseX, permutation);
        interp1 = interpolate(dot1, dot2, wz);

        dot1 = dotProduct(zGrid0, xGrid1, noiseZ, noiseX, permutation);
        dot2 = dotProduct(zGrid1, xGrid1, noiseZ, noiseX, permutation);
        interp2 = interpolate(dot1, dot2, wz);

        double noiseVal = interpolate(interp1, interp2, wx);
        total += noiseVal * amplitude;
        maxVal += amplitude;
    }
    total = total/maxVal;
    // apply terrain calcs
    total = (total + 1)/2;
    total = (int)floor(total * Terrain::TERRAIN_AMPLITUDE);
    heightMap[threadIdx.x] = total;
    return;
};

int* chunkHeightMapKernel(int chunkZ, int chunkX, int* permutation) {
    int* d_heightMap;
    int* d_permutation;
    size_t heightMapSize = sizeof(int)*Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH;
    int* heightMap = new int[Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH];
    hipMalloc(&d_heightMap, heightMapSize);
    size_t permutationSize = sizeof(int)*256;
    hipMalloc(&d_permutation, permutationSize);
    hipMemcpy(d_permutation, permutation, permutationSize, hipMemcpyHostToDevice);
    int block_width = Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH;
    dim3 dimBlock(block_width, 1, 1);
    dim3 dimGrid(1, 1, 1);
    chunkHeightMapKernel<<<dimGrid, dimBlock>>>(chunkZ, chunkX, d_heightMap, d_permutation);
    hipMemcpy(heightMap, d_heightMap, heightMapSize, hipMemcpyDeviceToHost);
    return heightMap;
}


// int main(int argc, char *argv[]) {
//     Terrain terrain(2022);

//     int* d_heightMap;
//     size_t heightMapSize = sizeof(int) * Terrain::CHUNK_WIDTH * Terrain::CHUNK_WIDTH;
//     int* heightMap = new int[Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH];

//     int* d_permutation;
//     size_t permutationSize = sizeof(int)*256;
    
//     int block_width = Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH;
//     dim3 dimBlock(block_width, 1, 1);
//     dim3 dimGrid(1, 1, 1);

//     // setup gpu timers
//     hipEvent_t start, stop;
//     float time;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);
//     hipEventRecord(start, 0);

//     // using constant memory for gradients and heightMap & shared memory
//     for (int i = 0; i < 1000 ; ++i) {
//         hipMalloc(&d_heightMap, heightMapSize);
//         hipMalloc(&d_permutation, permutationSize);
//         hipMemcpy(d_permutation, terrain.noise2D.permutation, permutationSize, hipMemcpyHostToDevice);
//         chunkHeightMapKernel<<<dimGrid, dimBlock>>>(0, 0, d_heightMap, d_permutation);
//         hipFree(d_heightMap);
//     }

//     // stop gpu timers
//     hipEventRecord(stop, 0);
//     hipEventSynchronize(stop); // after hipEventRecord
//     hipEventElapsedTime(&time, start, stop);
//     hipEventDestroy(start);
//     hipEventDestroy(stop);

//     printf("chunkHeightMapKernel time elapsed after 1000 kernel executions: %fms\n", time);
//     delete[] heightMap;
//     return 0;
// }
