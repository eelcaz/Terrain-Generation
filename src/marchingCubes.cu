#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <>
#include <vector>
#include <glm/glm.hpp>
#include <terrain_generator.h>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <time.h>
#include <chrono>

__constant__ unsigned int tris[256][16];
__constant__ unsigned int c2np[256];

#define CH 256
#define NCS 4
#define CW 16

__global__ void marchingCubesGPU(size_t* slices, GLfloat* vertices, float* chunks) {
    int k = threadIdx.x;
    int l = blockIdx.y;
    int m = blockIdx.x;
    int chunkA = l * 2 * NCS + m;
    //printf("%d\n", chunkA);
    int chunk = chunkA * CW * CW * CH;
    size_t index = slices[(CH) * chunkA + k];
    if (slices[(CH) * chunkA + k + 1] - index == 0) return;
    //printf("%d\n", index);
    //printf("%d\n", l);
    //printf("%ld\n", index);
    //printf("%d\n", chunks[0]);
    /*if (k == 0 && m == 0 && l == 0) {
        for (int i = 0; i < 256; i++) {
            for (int j = 0; j < 16; j++) {
                printf("%d ", tris[i][j]);
            }
            printf("\n");
        }
        printf("im alive\n");
    }*/
    for (int i = 0; i < CW - 1; i++) {
        for (int j = 0; j < CW - 1; j++) {
            //size_t curVoxel = 90 * (k * w * w + i * w + j);
            int b = 0;

            // flat array indexing for gpu returned chunks
            int k_ = k * CW * CW;
            int k_1 = k_ + CW * CW;
            int i_ = i * CW;
            int i_1 = i_ + CW;
            int j_ = j;
            int j_1 = j_ + 1;
            b += chunks[chunk+ k_ + i_1 + j_1] > Terrain::CAVE_INTENSITY; // v7
            b <<= 1;
            b += chunks[chunk+k_1 + i_1 + j_1] > Terrain::CAVE_INTENSITY; // v6
            b <<= 1;
            b += chunks[chunk+k_1 + i_ + j_1] > Terrain::CAVE_INTENSITY; // v5
            b <<= 1;
            b += chunks[chunk+k_ + i_ + j_1] > Terrain::CAVE_INTENSITY; // v4
            b <<= 1;
            b += chunks[chunk+k_ + i_1 + j_] > Terrain::CAVE_INTENSITY; // v3
            b <<= 1;
            b += chunks[chunk+k_1 + i_1 + j_] > Terrain::CAVE_INTENSITY; // v2
            b <<= 1;
            b += chunks[chunk+k_1 + i_ + j_] > Terrain::CAVE_INTENSITY; // v1
            b <<= 1;
            b += chunks[chunk+k_ + i_ + j_] > Terrain::CAVE_INTENSITY; // v0
            //printf("%d\n", b);
            unsigned int numTriangles = c2np[b];
            GLfloat edges[12][3] = {
                {i + 0.0f, j + 0.0f, k + 0.5f},    // e0
                {i + 0.5f, j + 0.0f, k + 1.0f},    // e1
                {i + 1.0f, j + 0.0f, k + 0.5f},    // e2
                {i + 0.5f, j + 0.0f, k + 0.0f},    // e3
                {i + 0.0f, j + 1.0f, k + 0.5f},    // e4
                {i + 0.5f, j + 1.0f, k + 1.0f},    // e5
                {i + 1.0f, j + 1.0f, k + 0.5f},    // e6
                {i + 0.5f, j + 1.0f, k + 0.0f},    // e7
                {i + 0.0f, j + 0.5f, k + 0.0f},    // e8
                {i + 0.0f, j + 0.5f, k + 1.0f},    // e9
                {i + 1.0f, j + 0.5f, k + 1.0f},    // e10
                {i + 1.0f, j + 0.5f, k + 0.0f}     // e11
            };
            int d = 1;

            //glm::vec3 grad(0, 0, 0);
            int k_d = k_ + d * CW * CW;
            int i_d = i_ + d * CW;
            int j_d = j_ + d;
            float gx = (float)chunks[chunk+k_ + i_ + j_d] - chunks[chunk+k_ + i_ + j_];
            float gy = (float)chunks[chunk+k_d + i_ + j_] - chunks[chunk+k_ + i_ + j_];
            float gz = (float)chunks[chunk+k_ + i_d + j_] - chunks[chunk+k_ + i_ + j_];
            //grad = -normalize(grad);
            //printf("%d\n", numTriangles);
            for (unsigned int iterate = 0; iterate < numTriangles; iterate++) {
                for (int ij = 0; ij < 3; ij++) {
                    auto curEdge = tris[b][iterate * 3 + ij];
                    auto px = edges[curEdge][1] + (CW - 1) * (m-NCS);
                    auto py = edges[curEdge][2];
                    auto pz = edges[curEdge][0] + (CW - 1) * (l-NCS);
                    vertices[index++] = px*2;
                    vertices[index++] = py*2;
                    vertices[index++] = pz*2;
                    vertices[index++] = -gx;
                    vertices[index++] = -gy;
                    vertices[index++] = -gz;
                }
            }
        }

        //__syncthreads();
    }
}

__global__ void getSlicesGPU(size_t* slices, float* chunks) {
    // represents the current chunk
    int k = threadIdx.x;
    int l = blockIdx.y;
    int m = blockIdx.x;
    //input1[tx] = input[tx];
    int i, j;
    int chunkA = l * 2 * NCS + m;
    //printf("%d\n", chunkA);
    int chunk = chunkA * CW * CW * CH;
    //slices[curSlice++] = num;
    int num = 0;
    for (i = 0; i < Terrain::CHUNK_WIDTH - 1; i++) {
        for (j = 0; j < Terrain::CHUNK_WIDTH - 1; j++) {
            int b = 0;

            // flat array indexing for gpu returned chunks
            int k_ = k * CW * CW;
            int k_1 = k_ + CW * CW;
            int i_ = i * CW;
            int i_1 = i_ + CW;
            int j_ = j;
            int j_1 = j_ + 1;
            b += chunks[chunk + k_ + i_1 + j_1] > Terrain::CAVE_INTENSITY; // v7
            b <<= 1;
            b += chunks[chunk + k_1 + i_1 + j_1] > Terrain::CAVE_INTENSITY; // v6
            b <<= 1;
            b += chunks[chunk + k_1 + i_ + j_1] > Terrain::CAVE_INTENSITY; // v5
            b <<= 1;
            b += chunks[chunk + k_ + i_ + j_1] > Terrain::CAVE_INTENSITY; // v4
            b <<= 1;
            b += chunks[chunk + k_ + i_1 + j_] > Terrain::CAVE_INTENSITY; // v3
            b <<= 1;
            b += chunks[chunk + k_1 + i_1 + j_] > Terrain::CAVE_INTENSITY; // v2
            b <<= 1;
            b += chunks[chunk + k_1 + i_ + j_] > Terrain::CAVE_INTENSITY; // v1
            b <<= 1;
            b += chunks[chunk + k_ + i_ + j_] > Terrain::CAVE_INTENSITY; // v0
            //printf("%d\n", b);
            unsigned int numTriangles = c2np[b];
            num += numTriangles * 18;
        }
    }
    if (k == 256) num = 0;
    __syncthreads();
    slices[chunkA * CH + k+1] = num;

}

void slicesKernel(size_t* slices, std::vector<float*> chunks, unsigned int triTable[256][16], unsigned int case_to_numpolys[256]) {
    float* chunks_host = (float*)calloc(chunks.size() * CW * CW * CH, sizeof(float));
    size_t index = 0;
    //thrust::device_vector<int> dummy_vec(chunks.size() * (CH)+1);
    for (int i = 0; i < chunks.size(); i++) {
        for (int j = 0; j < CW * CW * CH; j++) {
            chunks_host[index++] = chunks[i][j];
        }
    }

    float* chunks_device;
    size_t* slices_device;

    size_t chunks_size = chunks.size() * CW * CW * CH * sizeof(float);
    size_t slices_size = (chunks.size() * (CH)+1) * sizeof(size_t);

    hipMalloc((void**)&chunks_device, chunks_size);
    hipMalloc((void**)&slices_device, slices_size);

    //thrust::host_vector<size_t> h_data(chunks.size() * (CH)+1);
    //thrust::device_vector<size_t> d_data(chunks.size() * CH + 1);

    hipMemcpy(chunks_device, chunks_host, chunks_size, hipMemcpyHostToDevice);
    hipMemcpy(slices_device, slices, slices_size, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(c2np), case_to_numpolys, 256 * sizeof(unsigned int));
    hipMemcpyToSymbol(HIP_SYMBOL(tris), triTable, 256 * 16 * sizeof(unsigned int));

    dim3 blockDim, gridDim;
    gridDim = dim3(2 * NCS, 2 * NCS);
    blockDim = dim3(CH);
    //printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
      //  gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //thrust::fill(h_data.begin(), h_data.end(), 0);
    //thrust::copy(h_data.begin(), h_data.end(), d_data.begin());

    //std::cout << "warm up iteration " << d_data[0] << std::endl;
    //thrust::fill(d_data.begin(), d_data.end(), 0);
    //thrust::copy(d_data.begin(), d_data.end(), h_data.begin());
    //std::cout << "warm up iteration " << h_data[0] << std::endl;
    //thrust::fill(h_data.begin(), h_data.end(), 0);

    hipEventRecord(start);

    getSlicesGPU << <gridDim, blockDim >> > (slices_device, chunks_device);
    //printf("Device call:\t%s\n", hipGetErrorString(hipGetLastError()));

    //hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Marching Cubes Pt 1 (GPU): %f milliseconds\n", time);
    hipMemcpy(slices, slices_device, slices_size, hipMemcpyDeviceToHost);

    
    //thrust::device_vector<size_t> d_data(slices, slices + chunks.size() * (CH)+1);

    //for (int i = 0; i < 512; i++) {
      //  printf("%d, %d\n", i, slices[i]);
    //}
    /*for (int i = 0; i < (chunks.size() * (CH)+1); i++) {
        d_data[i] = slices[i];
    }*/

    //hipEventRecord(start);

    //thrust::inclusive_scan(d_data.begin(), d_data.end(), d_data.begin());

    //hipEventRecord(stop);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&time, start, stop);

    //printf("Marching Cubes Pt 1 Thrust (GPU): %f milliseconds\n", time);

    //thrust::copy(d_data.begin(), d_data.end(), slices);
    
    /*for (int i = 0; i < h_data.size(); i++) {
        slices[i] = h_data[i];
    }*/
    /*for (int i = 0; i < 512; i++) {
        printf("%d, %d\n", i, slices[i]);
    }*/
    //clock_t begin = clock();
    //hipEventRecord(start);
    std::chrono::time_point<std::chrono::system_clock> start2, end2;

    start2 = std::chrono::system_clock::now();

    size_t tracker = 0;
    for (int i = 0; i < (chunks.size() * (CH)+1); i++) {
        size_t temp = slices[i];
        slices[i] += tracker;
        tracker += temp;
    }
    end2 = std::chrono::system_clock::now();
    //hipEventRecord(stop);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&time, start, stop);
    //clock_t end = clock();
    //double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    //time_spent *= 1000;
    std::chrono::duration<double> elapsed_seconds = end2 - start2;
    printf("Marching Cubes Pt 1 Non-Thrust (CPU): %f milliseconds\n", elapsed_seconds.count() * 1000);
    /*for (int i = 0; i < 512; i++) {
        printf("%d, %d\n", i, slices[i]);
    }*/
    free(chunks_host);
    hipFree(slices_device);
    hipFree(chunks_device);
}

void marchingCubesKernel(size_t* slices, GLfloat* vertices_3D, std::vector<float*> chunks, size_t num, unsigned int triTable[256][16], unsigned int case_to_numpolys[256]) {
    float* chunks_host = (float*)calloc(chunks.size() * CW * CW * CH, sizeof(float));
    size_t index = 0;

    for (int i = 0; i < chunks.size(); i++) {
        for (int j = 0; j < CW * CW * CH; j++) {
            chunks_host[index++] = chunks[i][j];
        }
    }

    float* chunks_device;
    GLfloat* vertices_device;
    size_t* slices_device;

    size_t chunks_size = chunks.size() * CW * CW * CH * sizeof(float);
    size_t vertices_size = num * sizeof(GLfloat);
    size_t slices_size = (chunks.size() * (CH) +1)* sizeof(size_t);

    hipMalloc((void**)&chunks_device, chunks_size);
    hipMalloc((void**)&vertices_device, vertices_size);
    hipMalloc((void**)&slices_device, slices_size);

    hipMemcpy(chunks_device, chunks_host, chunks_size, hipMemcpyHostToDevice);
    hipMemcpy(vertices_device, vertices_3D, vertices_size, hipMemcpyHostToDevice);
    hipMemcpy(slices_device, slices, slices_size, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(c2np), case_to_numpolys, 256 * sizeof(unsigned int));
    hipMemcpyToSymbol(HIP_SYMBOL(tris), triTable, 256 * 16 * sizeof(unsigned int));

    dim3 blockDim, gridDim;
    gridDim = dim3(2 * NCS, 2 * NCS);
    blockDim = dim3(CH - 1);
    //printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
      //  gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
    
    hipEvent_t start2, stop2;
    float time2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    //printf("%d\n", NCS);
    marchingCubesGPU << <gridDim, blockDim >> > (slices_device, vertices_device, chunks_device);
    //printf("Device call:\t%s\n", hipGetErrorString(hipGetLastError()));

    //hipDeviceSynchronize();
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&time2, start2, stop2);
    hipMemcpy(vertices_3D, vertices_device, vertices_size, hipMemcpyDeviceToHost);

    printf("Marching Cubes Pt 2 (GPU): %f milliseconds\n", time2);
    free(chunks_host);
    //for (int i = 0; i < 512; i++) {
      //  printf("%d, %d\n", i, slices[i]);
    //}
    hipFree(vertices_device);
    hipFree(slices_device);
    hipFree(chunks_device);
}