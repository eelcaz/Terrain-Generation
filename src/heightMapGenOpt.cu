#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include "terrain_generator.h"


__device__ double interpolateOpt(double a, double b, double weight) {
    if (weight < 0) return a;
    if (weight > 1) return b;
    return (b - a) * ((weight * (weight * 6.0 - 15.0) + 10.0) * weight * weight * weight) + a;
};

__device__ double dotProductOpt(int GridZ, int GridX, double pz, double px, int* permutation) {
    // get the random vector on the gridPoint
    int randDir = permutation[(permutation[abs(GridZ) % 256] + abs(GridX)) % 256];
    double gradZ = cos((double)randDir);
    double gradX = sin((double)randDir);
    // get the offset vector from the grid point to the target point
    double offsetZ = pz-(double)GridZ;
    double offsetX = px-(double)GridX;

    return gradZ * offsetZ + gradX * offsetX;
};

__global__ void chunkHeightMapKernelOpt(int chunkZ, int chunkX, int* heightMap, int* permutation) {
    __shared__ float s_totals[64];
    int sectionSize = 64;
    int id = (threadIdx.x % sectionSize) + sectionSize*blockIdx.x;
    double offset = (double)1/(2*(Terrain::CHUNK_WIDTH-1));
    int _z = id / Terrain::CHUNK_WIDTH;
    int _x = id % Terrain::CHUNK_WIDTH;
    double z = (chunkZ + offset + (double)_z/(Terrain::CHUNK_WIDTH-1))/Terrain::TERRAIN_ZOOM;
    double x = (chunkX + offset + (double)_x/(Terrain::CHUNK_WIDTH-1))/Terrain::TERRAIN_ZOOM;
    double noiseZ, noiseX;

    // fbm iterations

    int i = threadIdx.x / sectionSize;

    double total = 0.0;
    double maxVal = 0;

    double amplitude = pow(0.58, (double) i);
    double frequency = pow(2.0, (double) i);

    noiseZ = z * frequency;
    noiseX = x * frequency;

    // noise calculations
    int zGrid0 = (int)floor(noiseZ);
    int xGrid0 = (int)floor(noiseX);
    int zGrid1 = zGrid0 + 1;
    int xGrid1 = xGrid0 + 1;

    // calculate weights
    double wz = noiseZ - (double)zGrid0;
    double wx = noiseX - (double)xGrid0;

    double dot1, dot2, interp1, interp2;
    dot1 = dotProductOpt(zGrid0, xGrid0, noiseZ, noiseX, permutation);
    dot2 = dotProductOpt(zGrid1, xGrid0, noiseZ, noiseX, permutation);
    interp1 = interpolateOpt(dot1, dot2, wz);

    dot1 = dotProductOpt(zGrid0, xGrid1, noiseZ, noiseX, permutation);
    dot2 = dotProductOpt(zGrid1, xGrid1, noiseZ, noiseX, permutation);
    interp2 = interpolateOpt(dot1, dot2, wz);

    double noiseVal = interpolateOpt(interp1, interp2, wx);

    double localTotal = noiseVal * amplitude;

    // add to final total
    // if (id >= 64) {
    //     printf("threadIdx: %d, blockIdx: %d id: %d\n", threadIdx.x, blockIdx.x, id);
    // }

    atomicAdd(&s_totals[id % 64], (float)localTotal);
    // printf("s_totals[%d]: %f\n", id%64, s_totals[id%64]);

    __syncthreads();
    // divide by maxVal
    if (i == 0) {
        for (int j = 0; j < 6; ++j) {
            maxVal += pow(0.58, (double) j);
        }

        total = s_totals[id % 64];
        // printf("id: %d, total: %f, s_totals: %f\n", id, total, s_totals[id % 64]);
        total /= maxVal;
        // printf("id: %d, total: %f, maxVal: %f\n", id, total, maxVal);
        total = (total + 1)/2;

        heightMap[id] = (int)floor((double)total * Terrain::TERRAIN_AMPLITUDE);
    }

    return;
};

// void setConstantPermutation(int* permutation) {
//     size_t permutationSize = sizeof(int)*256;
//     hipMemcpyToSymbol(HIP_SYMBOL(c_permutation), permutation, permutationSize);
// };

int* chunkHeightMapKernelOpt(int chunkZ, int chunkX, int* permutation) {
    int* d_heightMap;
    int* d_permutation;

    size_t heightMapSize = sizeof(int)*Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH;
    int* heightMap = new int[Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH];
    hipMalloc(&d_heightMap, heightMapSize);

    size_t permutationSize = sizeof(int)*256;
    hipMalloc(&d_permutation, permutationSize);
    hipMemcpy(d_permutation, permutation, permutationSize, hipMemcpyHostToDevice);

    int block_width = Terrain::CHUNK_WIDTH*(Terrain::CHUNK_WIDTH/4)*6;
    dim3 dimBlock(block_width, 1, 1);
    dim3 dimGrid(4, 1, 1);
    chunkHeightMapKernelOpt<<<dimGrid, dimBlock>>>(chunkZ, chunkX, d_heightMap, d_permutation);
    hipMemcpy(heightMap, d_heightMap, heightMapSize, hipMemcpyDeviceToHost);
    return heightMap;
}


int main(int argc, char *argv[]) {
    Terrain terrain(2022);

    int* d_heightMap;
    size_t heightMapSize = sizeof(int) * Terrain::CHUNK_WIDTH * Terrain::CHUNK_WIDTH;
    int* heightMap = new int[Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH];

    int* d_permutation;
    size_t permutationSize = sizeof(int)*256;

    int block_width = Terrain::CHUNK_WIDTH*(Terrain::CHUNK_WIDTH/4)*6;
    dim3 dimBlock(block_width, 1, 1);
    dim3 dimGrid(4, 1, 1);

    // setup gpu timers
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // using constant memory for gradients and heightMap & shared memory
    for (int i = 0; i < 1000 ; ++i) {
        hipMalloc(&d_heightMap, heightMapSize);
        hipMalloc(&d_permutation, permutationSize);
        hipMemcpy(d_permutation, terrain.noise2D.permutation, permutationSize, hipMemcpyHostToDevice);
        chunkHeightMapKernelOpt<<<dimGrid, dimBlock>>>(0, 0, d_heightMap, d_permutation);
        hipFree(d_heightMap);
    }

    // stop gpu timers
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); // after hipEventRecord
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("chunkHeightMapKernelOpt time elapsed after 1000 kernel executions: %fms\n", time);
    delete[] heightMap;
    return 0;
}
