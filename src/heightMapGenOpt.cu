#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include "terrain_generator.h"

__device__ __constant__ int c_permutation[256];

__device__ double interpolateOpt(double a, double b, double weight) {
    if (weight < 0) return a;
    if (weight > 1) return b;
    return (b - a) * ((weight * (weight * 6.0 - 15.0) + 10.0) * weight * weight * weight) + a;
};

__device__ double dotProductOpt(int GridZ, int GridX, double pz, double px) {
    // get the random vector on the gridPoint
    int randDir = c_permutation[(c_permutation[abs(GridZ) % 256] + abs(GridX)) % 256];
    double gradZ = cos((double)randDir);
    double gradX = sin((double)randDir);
    // get the offset vector from the grid point to the target point
    double offsetZ = pz-(double)GridZ;
    double offsetX = px-(double)GridX;

    return gradZ * offsetZ + gradX * offsetX;
};

__global__ void chunkHeightMapKernelOpt(int chunkZ, int chunkX, int* chunk) {
    double offset = (double)1/(2*(Terrain::CHUNK_WIDTH-1));
    int _z = threadIdx.x / Terrain::CHUNK_WIDTH;
    int _x = threadIdx.x % Terrain::CHUNK_WIDTH;
    double z = (chunkZ + offset + (double)_z/(Terrain::CHUNK_WIDTH-1))/Terrain::TERRAIN_ZOOM;
    double x = (chunkX + offset + (double)_x/(Terrain::CHUNK_WIDTH-1))/Terrain::TERRAIN_ZOOM;
    double noiseZ, noiseX;

    // fbm iterations
    int octaves = 6;
    double total = 0.0;
    double maxVal = 0;
    for (int i = 0; i < octaves; ++i) {
        double amplitude = pow(0.58, (double) i);
        double frequency = pow(2.0, (double) i);

        noiseZ = z * frequency;
        noiseX = x * frequency;

        // noise calculations
        int zGrid0 = (int)floor(noiseZ);
        int xGrid0 = (int)floor(noiseX);
        int zGrid1 = zGrid0 + 1;
        int xGrid1 = xGrid0 + 1;

        // calculate weights
        double wz = noiseZ - (double)zGrid0;
        double wx = noiseX - (double)xGrid0;

        double dot1, dot2, interp1, interp2;
        dot1 = dotProductOpt(zGrid0, xGrid0, noiseZ, noiseX);
        dot2 = dotProductOpt(zGrid1, xGrid0, noiseZ, noiseX);
        interp1 = interpolateOpt(dot1, dot2, wz);

        dot1 = dotProductOpt(zGrid0, xGrid1, noiseZ, noiseX);
        dot2 = dotProductOpt(zGrid1, xGrid1, noiseZ, noiseX);
        interp2 = interpolateOpt(dot1, dot2, wz);

        double noiseVal = interpolateOpt(interp1, interp2, wx);
        total += noiseVal * amplitude;
        maxVal += amplitude;
    }
    total = total/maxVal;
    // apply terrain calcs
    total = (total + 1)/2;
    total = (int)floor(total * Terrain::TERRAIN_AMPLITUDE);
    chunk[threadIdx.x] = total;
    return;
};

void setConstantPermutation(int* permutation) {
    size_t permutationSize = sizeof(int)*256;
    hipMemcpyToSymbol(HIP_SYMBOL(c_permutation), permutation, permutationSize);
};

int* chunkHeightMapKernelOpt(int chunkZ, int chunkX) {
    int* d_chunk;
    size_t chunkSize = sizeof(int)*Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH;
    int* chunk = new int[Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH];
    hipMalloc(&d_chunk, chunkSize);

    int block_width = Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH;
    dim3 dimBlock(block_width, 1, 1);
    dim3 dimGrid(1, 1, 1);
    chunkHeightMapKernelOpt<<<dimGrid, dimBlock>>>(chunkZ, chunkX, d_chunk);
    hipMemcpy(chunk, d_chunk, chunkSize, hipMemcpyDeviceToHost);
    return chunk;
}


// int main(int argc, char *argv[]) {
//     int* d_chunk;
//     size_t chunkSize = sizeof(int) * Terrain::CHUNK_WIDTH * Terrain::CHUNK_WIDTH;
//     chunk* chunk = new int[Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH];
//     hipMalloc(&d_chunk, chunkSize);
//     int block_width = Terrain::CHUNK_WIDTH*Terrain::CHUNK_WIDTH;
//     dim3 dimBlock(block_width, 1, 1);
//     dim3 dimGrid(1, 1, 1);
//     hipMemcpyToSymbol(HIP_SYMBOL("PERMUTATION"), &PERMUTATION, sizeof(PERMUTATION));
//     chunkHeightMapKernel<<<dimGrid, dimBlock>>>(0, 0, d_chunk);
//     hipMemcpy(chunk, d_chunk, chunkSize, hipMemcpyDeviceToHost);

//     for (int z = 0; z < Terrain::CHUNK_WIDTH; ++z) {
//         for (int x = 0; x < Terrain::CHUNK_WIDTH; ++x) {
//             std::cout << std::left << std::setw(12)
//                       << chunk[z*Terrain::CHUNK_WIDTH + x] << " ";
//         }
//         std::cout << "\n";
//     }
//     std::cout << std::endl;
//     delete[] chunk;
//     return 0;
// }
